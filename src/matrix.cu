#include "hip/hip_runtime.h"
#include "matrix.h"
const float th13 = 5.37;
hipComplex cuCexpf(hipComplex z) {

    float x = z.x;
    float y = z.y;
    float exp_x = expf(x);
    float cos_y = cosf(y);
    float sin_y = sinf(y);
    return make_hipComplex(exp_x * cos_y, exp_x * sin_y);
}

hipDoubleComplex cuCexp(hipDoubleComplex z) {

    double x = z.x;
    double y = z.y;
    double exp_x = expf(x);
    double cos_y = cosf(y);
    double sin_y = sinf(y);
    return make_hipDoubleComplex(exp_x * cos_y, exp_x * sin_y);
}

hipblasStatus_t gemm(hipblasHandle_t handle, int32_t M, int32_t K, int32_t N, float *d_A, float *d_B, float *d_C, float alpha, float beta) {
    /*
    d_A (MxK), d_B(KxN), d_C(MxN) row-major
    */ 
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    // float alpha = 1.0f;
    // float beta = 0.0f;
    return hipblasSgemm(handle,
                  transa, transb, 
                  N, M, K, 
                  &alpha, 
                  d_B, N,
                  d_A, K,
                  &beta,
                  d_C, N);
}

hipblasStatus_t gemm(hipblasHandle_t handle, int32_t M, int32_t K, int32_t N, double *d_A,
        double *d_B, double *d_C, double alpha, double beta) {
    /*
    d_A (MxK), d_B(KxN), d_C(MxN) row-major
    */ 
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    // float alpha = 1.0f;
    // float beta = 0.0f;
    return hipblasDgemm(handle,
                  transa, transb, 
                  N, M, K, 
                  &alpha, 
                  d_B, N,
                  d_A, K,
                  &beta,
                  d_C, N);
}

hipblasStatus_t gemm(hipblasHandle_t handle, int32_t M, int32_t K, int32_t N, hipComplex *d_A,
        hipComplex *d_B, hipComplex *d_C, hipComplex alpha, hipComplex beta) {
    /*
    d_A (MxK), d_B(KxN), d_C(MxN) row-major
    */ 
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    // float alpha = 1.0f;
    // float beta = 0.0f;
    return hipblasCgemm(handle,
                  transa, transb, 
                  N, M, K, 
                  &alpha, 
                  d_B, N,
                  d_A, K,
                  &beta,
                  d_C, N);
}

hipblasStatus_t gemm(hipblasHandle_t handle, int32_t M, int32_t K, int32_t N, hipDoubleComplex *d_A, hipDoubleComplex *d_B, hipDoubleComplex *d_C, hipDoubleComplex alpha, hipDoubleComplex beta) {
    /*
    d_A (MxK), d_B(KxN), d_C(MxN) row-major
    */ 
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    // float alpha = 1.0f;
    // float beta = 0.0f;
    return hipblasZgemm(handle,
                  transa, transb, 
                  N, M, K, 
                  &alpha, 
                  d_B, N,
                  d_A, K,
                  &beta,
                  d_C, N);
}

void solve(hipsolverHandle_t handle,float *d_A, float *d_B, int m) {
    using data_type = float;
    const int64_t lda = m;
    const int64_t ldb = m;
    
    int64_t *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr;     /* error info */

    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    void *d_work = nullptr;              /* device workspace for getrf */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void *h_work = nullptr;              /* host workspace for getrf */

    const int algo = 0;
    /* Create advanced params */
    hipsolverDnParams_t params;
    hipsolverDnCreateParams(&params);
    if (algo == 0) {
        hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0);
    } else {
        hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_1);
    }
    hipsolverDnXgetrf_bufferSize(handle, params, m, m, traits<data_type>::cuda_data_type, d_A,
                                    lda, traits<data_type>::cuda_data_type, &workspaceInBytesOnDevice,
                                    &workspaceInBytesOnHost);
    hipMalloc(reinterpret_cast<void **>(&d_work), workspaceInBytesOnDevice);
    hipMalloc(reinterpret_cast<void **>(&d_Ipiv), sizeof(int64_t) * m);
    hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int));
    if (0 < workspaceInBytesOnHost) {
        h_work = reinterpret_cast<void *>(malloc(workspaceInBytesOnHost));
        if (h_work == nullptr) {
            throw std::runtime_error("Error: h_work not allocated.");
        }
    }
    /* step 4: LU factorization */

    hipsolverDnXgetrf(handle, params, m, m, traits<data_type>::cuda_data_type,
                                        d_A, lda, d_Ipiv, traits<data_type>::cuda_data_type, d_work,
                                        workspaceInBytesOnDevice, h_work, workspaceInBytesOnHost, d_info);
    hipsolverDnXgetrs(handle, params, HIPBLAS_OP_N, m, m, /* nrhs */
                                        traits<data_type>::cuda_data_type, d_A, lda, d_Ipiv,
                                        traits<data_type>::cuda_data_type, d_B, ldb, d_info);
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_Ipiv));
    CUDA_CHECK(hipFree(d_work));
    free(h_work);
}


void solve(hipsolverHandle_t handle, double *d_A, double *d_B, int m) {
    using data_type = double;
    const int64_t lda = m;
    const int64_t ldb = m;

    int64_t *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr;     /* error info */

    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    void *d_work = nullptr;              /* device workspace for getrf */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void *h_work = nullptr;              /* host workspace for getrf */

    const int algo = 0;
    /* Create advanced params */
    hipsolverDnParams_t params;
    hipsolverDnCreateParams(&params);
    if (algo == 0) {
        hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0);
    } else {
        hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_1);
    }
    hipsolverDnXgetrf_bufferSize(handle, params, m, m, traits<data_type>::cuda_data_type, d_A,
                                    lda, traits<data_type>::cuda_data_type, &workspaceInBytesOnDevice,
                                    &workspaceInBytesOnHost);
    hipMalloc(reinterpret_cast<void **>(&d_work), workspaceInBytesOnDevice);
    hipMalloc(reinterpret_cast<void **>(&d_Ipiv), sizeof(int64_t) * m);
    hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int));
    if (0 < workspaceInBytesOnHost) {
        h_work = reinterpret_cast<void *>(malloc(workspaceInBytesOnHost));
        if (h_work == nullptr) {
            throw std::runtime_error("Error: h_work not allocated.");
        }
    }
    /* step 4: LU factorization */

    hipsolverDnXgetrf(handle, params, m, m, traits<data_type>::cuda_data_type,
                                        d_A, lda, d_Ipiv, traits<data_type>::cuda_data_type, d_work,
                                        workspaceInBytesOnDevice, h_work, workspaceInBytesOnHost, d_info);
    hipsolverDnXgetrs(handle, params, HIPBLAS_OP_N, m, m, /* nrhs */
                                        traits<data_type>::cuda_data_type, d_A, lda, d_Ipiv,
                                        traits<data_type>::cuda_data_type, d_B, ldb, d_info);
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_Ipiv));
    CUDA_CHECK(hipFree(d_work));
    free(h_work);
}

void solve(hipsolverHandle_t handle, hipComplex *d_A, hipComplex *d_B, int m) {
    using data_type = hipComplex;
    const int64_t lda = m;
    const int64_t ldb = m;

    int64_t *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr;     /* error info */

    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    void *d_work = nullptr;              /* device workspace for getrf */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void *h_work = nullptr;              /* host workspace for getrf */

    const int algo = 0;
    /* Create advanced params */
    hipsolverDnParams_t params;
    hipsolverDnCreateParams(&params);
    if (algo == 0) {
        hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0);
    } else {
        hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_1);
    }
    hipsolverDnXgetrf_bufferSize(handle, params, m, m, traits<data_type>::cuda_data_type, d_A,
                                    lda, traits<data_type>::cuda_data_type, &workspaceInBytesOnDevice,
                                    &workspaceInBytesOnHost);
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), workspaceInBytesOnDevice));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Ipiv), sizeof(int64_t) * m));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    if (0 < workspaceInBytesOnHost) {
        h_work = reinterpret_cast<void *>(malloc(workspaceInBytesOnHost));
        if (h_work == nullptr) {
            throw std::runtime_error("Error: h_work not allocated.");
        }
    }
    /* step 4: LU factorization */
    CUSOLVER_CHECK(hipsolverDnXgetrf(handle, params, m, m, traits<data_type>::cuda_data_type,
                                        d_A, lda, d_Ipiv, traits<data_type>::cuda_data_type, d_work,
                                        workspaceInBytesOnDevice, h_work, workspaceInBytesOnHost, d_info));
    CUSOLVER_CHECK(hipsolverDnXgetrs(handle, params, HIPBLAS_OP_N, m, m, /* nrhs */
                                        traits<data_type>::cuda_data_type, d_A, lda, d_Ipiv,
                                        traits<data_type>::cuda_data_type, d_B, ldb, d_info));
    hipDeviceSynchronize();

    CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
    CUDA_CHECK(hipFree(d_Ipiv));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));
    free(h_work);
}


void solve(hipsolverHandle_t handle, hipDoubleComplex *d_A, hipDoubleComplex *d_B, int m) {
    using data_type = hipDoubleComplex;
    const int64_t lda = m;
    const int64_t ldb = m;

    int64_t *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr;     /* error info */

    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    void *d_work = nullptr;              /* device workspace for getrf */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void *h_work = nullptr;              /* host workspace for getrf */

    const int algo = 0;
    /* Create advanced params */
    hipsolverDnParams_t params;
    hipsolverDnCreateParams(&params);
    if (algo == 0) {
        hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0);
    } else {
        hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_1);
    }
    hipsolverDnXgetrf_bufferSize(handle, params, m, m, traits<data_type>::cuda_data_type, d_A,
                                    lda, traits<data_type>::cuda_data_type, &workspaceInBytesOnDevice,
                                    &workspaceInBytesOnHost);
    hipMalloc(reinterpret_cast<void **>(&d_work), workspaceInBytesOnDevice);
    hipMalloc(reinterpret_cast<void **>(&d_Ipiv), sizeof(int64_t) * m);
    hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int));
    if (0 < workspaceInBytesOnHost) {
        h_work = reinterpret_cast<void *>(malloc(workspaceInBytesOnHost));
        if (h_work == nullptr) {
            throw std::runtime_error("Error: h_work not allocated.");
        }
    }
    /* step 4: LU factorization */

    hipsolverDnXgetrf(handle, params, m, m, traits<data_type>::cuda_data_type,
                                        d_A, lda, d_Ipiv, traits<data_type>::cuda_data_type, d_work,
                                        workspaceInBytesOnDevice, h_work, workspaceInBytesOnHost, d_info);
    hipsolverDnXgetrs(handle, params, HIPBLAS_OP_N, m, m, /* nrhs */
                                        traits<data_type>::cuda_data_type, d_A, lda, d_Ipiv,
                                        traits<data_type>::cuda_data_type, d_B, ldb, d_info);
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_Ipiv));
    CUDA_CHECK(hipFree(d_work));
    free(h_work);
}


// void solve(hipsolverHandle_t handle,float *d_A, float *d_B, int m, void *d_work, void *h_work, int64_t *d_Ipiv) {
//     using data_type = float;
//     const int64_t lda = m;
//     const int64_t ldb = m;

//     int64_t *d_Ipiv = nullptr; /* pivoting sequence */
//     int *d_info = nullptr;     /* error info */

//     size_t workspaceInBytesOnDevice = 0; /* size of workspace */
//     void *d_work = nullptr;              /* device workspace for getrf */
//     size_t workspaceInBytesOnHost = 0;   /* size of workspace */
//     void *h_work = nullptr;              /* host workspace for getrf */

//     const int algo = 0;
//     /* Create advanced params */
//     hipsolverDnParams_t params;
//     hipsolverDnCreateParams(&params);
//     if (algo == 0) {
//         hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0);
//     } else {
//         hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_1);
//     }
//     hipsolverDnXgetrf_bufferSize(handle, params, m, m, traits<data_type>::cuda_data_type, d_A,
//                                     lda, traits<data_type>::cuda_data_type, &workspaceInBytesOnDevice,
//                                     &workspaceInBytesOnHost);
//     hipMalloc(reinterpret_cast<void **>(&d_work), workspaceInBytesOnDevice);
//     hipMalloc(reinterpret_cast<void **>(&d_Ipiv), sizeof(int64_t) * m);
//     hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int));
//     if (0 < workspaceInBytesOnHost) {
//         h_work = reinterpret_cast<void *>(malloc(workspaceInBytesOnHost));
//         if (h_work == nullptr) {
//             throw std::runtime_error("Error: h_work not allocated.");
//         }
//     }
//     /* step 4: LU factorization */

//     hipsolverDnXgetrf(handle, params, m, m, traits<data_type>::cuda_data_type,
//                                         d_A, lda, d_Ipiv, traits<data_type>::cuda_data_type, d_work,
//                                         workspaceInBytesOnDevice, h_work, workspaceInBytesOnHost, d_info);
//     hipsolverDnXgetrs(handle, params, HIPBLAS_OP_N, m, m, /* nrhs */
//                                         traits<data_type>::cuda_data_type, d_A, lda, d_Ipiv,
//                                         traits<data_type>::cuda_data_type, d_B, ldb, d_info);
//     CUDA_CHECK(hipFree(d_info));
//     // CUDA_CHECK(hipFree(d_Ipiv));
//     // CUDA_CHECK(hipFree(d_work));
//     // free(h_work);
//     hipsolverDnDestroyParams(params);

// }
